#include "hip/hip_runtime.h"

#include <stdio.h>#include<malloc.h>
float cuda_malloc_test(int size, bool up)
{ hipEvent_t start, stop; 
int *a, *dev_a; 
float elapseTime; 
hipEventCreate(&start); 
hipEventCreate(&stop); 
a = (int*)malloc(size*sizeof(*a)); 
hipMalloc((void**)&dev_a, size*sizeof(*dev_a)); 
hipEventRecord(start, 0); 
for (size_t i = 0; i < 100; i++) 
{  if (up)   
   hipMemcpy(dev_a, a, size*sizeof(*dev_a), hipMemcpyHostToDevice);  
   else  {   hipMemcpy(a, dev_a, size*sizeof(*dev_a), hipMemcpyDeviceToHost); 
   }
 }
 hipEventRecord(stop, 0); 
 hipEventSynchronize(stop); 
 hipEventElapsedTime(&elapseTime, start, stop); 
 free(a); 
 hipFree(dev_a); 
 hipEventDestroy(start); 
 hipEventDestroy(stop);
 return elapseTime;
}


float cuda_host_alloc_test(int size, bool up)
{ hipEvent_t start, stop; 
int *a, *dev_a; 
float elapseTime; 
hipEventCreate(&start); 
hipEventCreate(&stop); 
hipHostAlloc((void**)&a,size*sizeof(*a),hipHostMallocDefault);  
hipMalloc((void**)&dev_a, size*sizeof(*dev_a)); 
hipEventRecord(start, 0); 
for (size_t i = 0; i < 100; i++) 
{  if (up)   
   hipMemcpy(dev_a, a, size*sizeof(*dev_a), hipMemcpyHostToDevice);  
	 else  {   
	 hipMemcpy(a, dev_a, size*sizeof(*dev_a), hipMemcpyDeviceToHost);
  }
 } 
 hipEventRecord(stop, 0); 
 hipEventSynchronize(stop); 
 hipEventElapsedTime(&elapseTime, start, stop);
 hipHostFree(a); 
 hipFree(dev_a); 
 hipEventDestroy(start); 
 hipEventDestroy(stop);
 return elapseTime;
 }
#define SIZE (10*1024*1024)

int main()
{ float elapsedTime; 
  float MB = (float)100 * SIZE*sizeof(int) / 1024 / 1024; 
	//elapsedTime = cuda_host_alloc_test(SIZE, false); 
	elapsedTime = cuda_malloc_test(SIZE, false); 
	printf("Time using cudamalloc:%3.1fms\n", elapsedTime); 
	printf("\tMB/s during copy up: %3.1f\n", MB / (elapsedTime / 1000));
}




